#include "hip/hip_runtime.h"
#include "fluid_simulation.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

// Constructor
FluidSimulation::FluidSimulation(int width, int height, int depth) {
    params.width = width;
    params.height = height;
    params.depth = depth;
    params.dt = 0.1f;
    params.visc = 0.0f;
    params.diff = 0.0f;
    
    allocateMemory();
}

// Destructor
FluidSimulation::~FluidSimulation() {
    freeMemory();
}

// Memory allocation
void FluidSimulation::allocateMemory() {
    size_t size = params.width * params.height * params.depth * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_s, size));
    CUDA_CHECK(hipMalloc(&d_density, size));

    CUDA_CHECK(hipMalloc(&d_Vx, size));
    CUDA_CHECK(hipMalloc(&d_Vy, size));
    CUDA_CHECK(hipMalloc(&d_Vz, size));
    
    CUDA_CHECK(hipMalloc(&d_Vx0, size));
    CUDA_CHECK(hipMalloc(&d_Vy0, size));
    CUDA_CHECK(hipMalloc(&d_Vz0, size));
    
    resetFields();
}

// Memory cleanup
void FluidSimulation::freeMemory() {
    CUDA_CHECK(hipFree(d_s));
    CUDA_CHECK(hipFree(d_density));

    CUDA_CHECK(hipFree(d_Vx));
    CUDA_CHECK(hipFree(d_Vy));
    CUDA_CHECK(hipFree(d_Vz));
    
    CUDA_CHECK(hipFree(d_Vx0));
    CUDA_CHECK(hipFree(d_Vy0));
    CUDA_CHECK(hipFree(d_Vz0));
}

// Reset all fields to zero
void FluidSimulation::resetFields() {
    size_t size = params.width * params.height * params.depth * sizeof(float);

    CUDA_CHECK(hipMemset(d_s, 0, size));
    CUDA_CHECK(hipMemset(d_density, 0, size));

    CUDA_CHECK(hipMemset(d_Vx, 0, size));
    CUDA_CHECK(hipMemset(d_Vy, 0, size));
    CUDA_CHECK(hipMemset(d_Vz, 0, size));
    
    CUDA_CHECK(hipMemset(d_Vx0, 0, size));
    CUDA_CHECK(hipMemset(d_Vy0, 0, size));
    CUDA_CHECK(hipMemset(d_Vz0, 0, size));
}

// Initialize simulation
void FluidSimulation::initialize() {
    resetFields();
}

// Main simulation step
void FluidSimulation::step() {
    // TODO: Implement the main simulation steps:
    // 1. Velocity step
    // 2. Density step
    // 3. Pressure solve
    // 4. Advection
}

int FluidSimulation::IX(int x, int y, int z) {
    return (z * params.height + y) * params.width + x;
}

// CUDA kernel for adding velocity
__global__ void addVelocityKernel(float* d_Vx, float* d_Vy, float* d_Vz, 
                                int x, int y, int z, float vx, float vy, float vz,
                                int width, int height, int depth) {
    int idx = (z * height + y) * width + x;
    if (x >= 0 && x < width && y >= 0 && y < height && z >= 0 && z < depth) {
        d_Vx[idx] += vx;
        d_Vy[idx] += vy;
        d_Vz[idx] += vz;
    }
}

// CUDA kernel for adding density
__global__ void addDensityKernel(float* d_density, int x, int y, int z, 
                               float amount, int width, int height, int depth) {
    int idx = (z * height + y) * width + x;
    if (x >= 0 && x < width && y >= 0 && y < height && z >= 0 && z < depth) {
        d_density[idx] += amount;
    }
}

// Get density field as host vector
std::vector<float> FluidSimulation::getDensityFieldHost() {
    size_t size = params.width * params.height * params.depth;
    std::vector<float> host_data(size);
    CUDA_CHECK(hipMemcpy(host_data.data(), d_density, size * sizeof(float), hipMemcpyDeviceToHost));
    return host_data;
}

// Get velocity field as host vector
std::vector<float> FluidSimulation::getVelocityFieldHost(int component) {
    size_t size = params.width * params.height * params.depth;
    std::vector<float> host_data(size);
    
    float* d_field;
    switch(component) {
        case 0: d_field = d_Vx; break;
        case 1: d_field = d_Vy; break;
        case 2: d_field = d_Vz; break;
        default: return std::vector<float>(); // Return empty vector for invalid component
    }
    
    CUDA_CHECK(hipMemcpy(host_data.data(), d_field, size * sizeof(float), hipMemcpyDeviceToHost));
    return host_data;
}

// Add velocity at a point (using CUDA kernel)
void FluidSimulation::addVelocity(int x, int y, int z, float vx, float vy, float vz) {
    if (x < 0 || x >= params.width || y < 0 || y >= params.height || z < 0 || z >= params.depth) return;
    
    dim3 block(1);
    dim3 grid(1);
    addVelocityKernel<<<grid, block>>>(d_Vx, d_Vy, d_Vz, x, y, z, vx, vy, vz,
                                      params.width, params.height, params.depth);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Add density at a point (using CUDA kernel)
void FluidSimulation::addDensity(int x, int y, int z, float amount) {
    if (x < 0 || x >= params.width || y < 0 || y >= params.height || z < 0 || z >= params.depth) return;
    
    dim3 block(1);
    dim3 grid(1);
    addDensityKernel<<<grid, block>>>(d_density, x, y, z, amount,
                                     params.width, params.height, params.depth);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Get current density field
float* FluidSimulation::getDensityField() {
    return d_density;
}
