#include "fluid_simulation.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

// Constructor
FluidSimulation::FluidSimulation(int width, int height, int depth) {
    params.width = width;
    params.height = height;
    params.depth = depth;
    params.dt = 0.1f;
    params.viscosity = 0.0f;
    params.diffusion = 0.0f;
    
    allocateMemory();
}

// Destructor
FluidSimulation::~FluidSimulation() {
    freeMemory();
}

// Memory allocation
void FluidSimulation::allocateMemory() {
    size_t size = params.width * params.height * params.depth * sizeof(float);
    
    CUDA_CHECK(hipMalloc(&d_velocityX, size));
    CUDA_CHECK(hipMalloc(&d_velocityY, size));
    CUDA_CHECK(hipMalloc(&d_velocityZ, size));
    CUDA_CHECK(hipMalloc(&d_density, size));
    CUDA_CHECK(hipMalloc(&d_prevVelocityX, size));
    CUDA_CHECK(hipMalloc(&d_prevVelocityY, size));
    CUDA_CHECK(hipMalloc(&d_prevVelocityZ, size));
    CUDA_CHECK(hipMalloc(&d_prevDensity, size));
    
    resetFields();
}

// Memory cleanup
void FluidSimulation::freeMemory() {
    CUDA_CHECK(hipFree(d_velocityX));
    CUDA_CHECK(hipFree(d_velocityY));
    CUDA_CHECK(hipFree(d_velocityZ));
    CUDA_CHECK(hipFree(d_density));
    CUDA_CHECK(hipFree(d_prevVelocityX));
    CUDA_CHECK(hipFree(d_prevVelocityY));
    CUDA_CHECK(hipFree(d_prevVelocityZ));
    CUDA_CHECK(hipFree(d_prevDensity));
}

// Reset all fields to zero
void FluidSimulation::resetFields() {
    size_t size = params.width * params.height * params.depth * sizeof(float);
    CUDA_CHECK(hipMemset(d_velocityX, 0, size));
    CUDA_CHECK(hipMemset(d_velocityY, 0, size));
    CUDA_CHECK(hipMemset(d_velocityZ, 0, size));
    CUDA_CHECK(hipMemset(d_density, 0, size));
    CUDA_CHECK(hipMemset(d_prevVelocityX, 0, size));
    CUDA_CHECK(hipMemset(d_prevVelocityY, 0, size));
    CUDA_CHECK(hipMemset(d_prevVelocityZ, 0, size));
    CUDA_CHECK(hipMemset(d_prevDensity, 0, size));
}

// Initialize simulation
void FluidSimulation::initialize() {
    resetFields();
}

// Main simulation step
void FluidSimulation::step() {
    // TODO: Implement the main simulation steps:
    // 1. Velocity step
    // 2. Density step
    // 3. Pressure solve
    // 4. Advection
}

// Add velocity at a point
void FluidSimulation::addVelocity(int x, int y, int z, float vx, float vy, float vz) {
    if (x < 0 || x >= params.width || y < 0 || y >= params.height || z < 0 || z >= params.depth) return;
    int index = (z * params.height + y) * params.width + x;
    CUDA_CHECK(hipMemcpy(d_prevVelocityX + index, &vx, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_prevVelocityY + index, &vy, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_prevVelocityZ + index, &vz, sizeof(float), hipMemcpyHostToDevice));
}

// Add density at a point
void FluidSimulation::addDensity(int x, int y, int z, float amount) {
    if (x < 0 || x >= params.width || y < 0 || y >= params.height || z < 0 || z >= params.depth) return;
    int index = (z * params.height + y) * params.width + x;
    CUDA_CHECK(hipMemcpy(d_prevDensity + index, &amount, sizeof(float), hipMemcpyHostToDevice));
}

// Get current density field
float* FluidSimulation::getDensityField() {
    return d_density;
} 